#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include <library_daq.h>

// CUDA = Computer Device Unified Architecture

__global__ void kernel_correct_times(unsigned int *ct);




//
// main code
//

int main(int argc, const char **argv)
{


  /////////////////////
  // initialise card //
  /////////////////////
  findCudaDevice(argc, argv);


  // initialise CUDA timing
  bool use_timing = true;
  if( use_timing ){
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }
  hipEventCreate(&total_start);
  hipEventCreate(&total_stop);
  float elapsed_parameters, elapsed_pmts, elapsed_detector, elapsed_vertices,
    elapsed_threads, elapsed_tof, elapsed_memory_tofs_dev, elapsed_tofs_copy_dev,
    elapsed_input, elapsed_memory_dev, elapsed_copy_dev, elapsed_kernel, 
    elapsed_threads_candidates, elapsed_candidates_memory_dev, elapsed_candidates_kernel,
    elapsed_candidates_copy_host, elapsed_coalesce, elapsed_gates, elapsed_free, elapsed_total,
    elapsed_tofs_free, elapsed_reset;
  bool use_verbose = true;


  ////////////////////
  // inspect device //
  ////////////////////
  print_gpu_properties();



  ///////////////////////
  // define parameters //
  ///////////////////////
  if( use_timing )
    start_c_clock();
  distance_between_vertices = 500.; // cm
  time_step_size  = 10; // ns
  threshold_number_of_pmts = 45;
  coalesce_time = 500.; // ns
  trigger_gate_up = 950.0; // ns
  trigger_gate_down = -400.0 -200; // ns
  output_txt = false;
  if( use_verbose ){
    printf(" --- user parameters \n");
    printf(" distance between test vertices = %f cm \n", distance_between_vertices);
    printf(" time step size = %d ns \n", time_step_size);
    printf(" threshold_number_of_pmts = %d \n", threshold_number_of_pmts);
    printf(" coalesce_time = %f ns \n", coalesce_time);
    printf(" trigger_gate_up = %f ns \n", trigger_gate_up);
    printf(" trigger_gate_down = %f ns \n", trigger_gate_down);
  }
  if( use_timing )
    elapsed_parameters = stop_c_clock();




  ////////////////
  // read PMTs  //
  ////////////////
  if( use_timing )
    start_c_clock();
  if( !read_the_pmts() ) return 0;
  if( use_timing )
    elapsed_pmts = stop_c_clock();


  /////////////////////
  // read detector ////
  /////////////////////
  if( use_timing )
    start_c_clock();
  if( !read_the_detector() ) return 0;
  if( use_timing )
    elapsed_detector = stop_c_clock();




  ////////////////////////
  // make test vertices //
  ////////////////////////
  if( use_timing )
    start_c_clock();
  make_test_vertices();
  if( use_timing )
    elapsed_vertices = stop_c_clock();



  //////////////////////////////
  // table of times_of_flight //
  //////////////////////////////
  if( use_timing )
    start_c_clock();
  make_table_of_tofs();
  if( use_timing )
    elapsed_tof = stop_c_clock();

  if( use_timing )
    start_cuda_clock();
  allocate_tofs_memory_on_device();
  if( use_timing )
    elapsed_memory_tofs_dev = stop_cuda_clock();


  if( use_timing )
    start_cuda_clock();
  fill_tofs_memory_on_device();
  if( use_timing )
    elapsed_tofs_copy_dev = stop_cuda_clock();


  ////////////////
  // read input //
  ////////////////
  if( use_timing )
    start_c_clock();
  if( !read_the_input() ) return 0;
  if( use_timing )
    elapsed_input = stop_c_clock();
  

  allocate_candidates_memory_on_host();


  ////////////////////////////////////////////////
  // set number of blocks and threads per block //
  ////////////////////////////////////////////////
  if( use_timing )
    start_c_clock();
  if( !setup_threads_for_tof() ) return 0;
  if( use_timing )
    elapsed_threads = stop_c_clock();


  start_total_cuda_clock();
  ///////////////////////////////
  // allocate memory on device //
  ///////////////////////////////
  if( use_timing )
    start_cuda_clock();
  allocate_correct_memory_on_device();
  if( use_timing )
    elapsed_memory_dev = stop_cuda_clock();


  //////////////////////////////////////
  // copy input into device variables //
  //////////////////////////////////////
  if( use_timing )
    start_cuda_clock();
  fill_correct_memory_on_device();
  if( use_timing )
    elapsed_copy_dev = stop_cuda_clock();



  ////////////////////
  // execute kernel //
  ////////////////////
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- execute kernel \n");
  kernel_correct_times<<<number_of_kernel_blocks,number_of_threads_per_block>>>(device_n_pmts_per_time_bin);
  getLastCudaError("correct_kernel execution failed\n");
  if( use_timing )
    elapsed_kernel = stop_cuda_clock();



  /////////////////////////////////////
  // find candidates above threshold //
  /////////////////////////////////////
  if( use_timing )
    start_c_clock();
  if( !setup_threads_to_find_candidates() ) return 0;
  if( use_timing )
    elapsed_threads_candidates = stop_c_clock();


  if( use_timing )
    start_cuda_clock();
  allocate_candidates_memory_on_device();
  if( use_timing )
    elapsed_candidates_memory_dev = stop_cuda_clock();

  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- execute candidates kernel \n");
  kernel_find_vertex_with_max_npmts_in_timebin<<<number_of_kernel_blocks,number_of_threads_per_block>>>(device_n_pmts_per_time_bin, device_max_number_of_pmts_in_time_bin, device_vertex_with_max_n_pmts);
  getLastCudaError("candidates_kernel execution failed\n");
  if( use_timing )
    elapsed_candidates_kernel = stop_cuda_clock();

  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- copy candidates from device to host \n");
  checkCudaErrors(hipMemcpy(host_max_number_of_pmts_in_time_bin,
			     device_max_number_of_pmts_in_time_bin,
			     n_time_bins*sizeof(unsigned int),
			     hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(host_vertex_with_max_n_pmts,
			     device_vertex_with_max_n_pmts,
			     n_time_bins*sizeof(unsigned int),
			     hipMemcpyDeviceToHost));
  if( use_timing )
    elapsed_candidates_copy_host = stop_cuda_clock();

  for(unsigned int time_bin = 0; time_bin<n_time_bins - 1; time_bin++){ // loop over time bins
    // n_time_bins - 1 as we are checking the i and i+1 at the same time
    
    if(host_max_number_of_pmts_in_time_bin[time_bin] > threshold_number_of_pmts) {

      if( use_verbose )
	printf(" time %f vertex (%f, %f, %f) npmts %d \n", (time_bin + 2)*time_step_size - time_offset, vertex_x[host_vertex_with_max_n_pmts[time_bin]], vertex_y[host_vertex_with_max_n_pmts[time_bin]], vertex_z[host_vertex_with_max_n_pmts[time_bin]], host_max_number_of_pmts_in_time_bin[time_bin]);

      candidate_trigger_pair_vertex_time.push_back(std::make_pair(host_vertex_with_max_n_pmts[time_bin],time_bin+2));
      candidate_trigger_npmts_in_time_bin.push_back(host_max_number_of_pmts_in_time_bin[time_bin]);
    }

  }

  if( use_verbose )
    printf(" n candidates: %d \n", candidate_trigger_pair_vertex_time.size());





  ///////////////////////
  // coalesce triggers //
  ///////////////////////
  if( use_timing )
    start_cuda_clock();
  coalesce_triggers();
  if( use_timing )
    elapsed_coalesce = stop_cuda_clock();




  //////////////////////////////////
  // separate triggers into gates //
  //////////////////////////////////
  if( use_timing )
    start_cuda_clock();
  separate_triggers_into_gates();
  if( use_timing )
    elapsed_gates = stop_cuda_clock();




  // deallocate all memory 
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- deallocate memory \n");
  free_event_memories();
  if( use_timing )
    elapsed_free = stop_cuda_clock();

  elapsed_total = stop_total_cuda_clock();

  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- deallocate tofs memory \n");
  free_global_memories();
  if( use_timing )
    elapsed_tofs_free = stop_cuda_clock();


  // CUDA exit -- needed to flush the buffer which holds printf from each thread
  if( use_timing )
    start_cuda_clock();
  if( use_verbose )
    printf(" --- reset device \n");
  //  hipDeviceReset();
  if( use_timing )
    elapsed_reset = stop_cuda_clock();


  if( use_timing ){
    printf(" user parameters time : %f ms \n", elapsed_parameters);
    printf(" read pmts execution time : %f ms \n", elapsed_pmts);
    printf(" read detector execution time : %f ms \n", elapsed_detector);
    printf(" make test vertices execution time : %f ms \n", elapsed_vertices);
    printf(" setup threads execution time : %f ms \n", elapsed_threads);
    printf(" setup threads candidates execution time : %f ms \n", elapsed_threads_candidates);
    printf(" make table of tofs execution time : %f ms \n", elapsed_tof);
    printf(" read input execution time : %f ms \n", elapsed_input);
    printf(" allocate tofs memory on device execution time : %f ms \n", elapsed_memory_tofs_dev);
    printf(" fill tofs memory on device execution time : %f ms \n", elapsed_tofs_copy_dev);
    printf(" deallocate tofs memory execution time : %f ms \n", elapsed_tofs_free);
    printf(" device reset execution time : %f ms \n", elapsed_reset);
    printf(" allocate memory on device execution time : %f ms (%f) \n", elapsed_memory_dev, elapsed_memory_dev/elapsed_total);
    printf(" fill memory on device execution time : %f ms (%f) \n", elapsed_copy_dev, elapsed_copy_dev/elapsed_total);
    printf(" correct kernel execution time : %f ms (%f) \n", elapsed_kernel, elapsed_kernel/elapsed_total);
    printf(" allocate candidates memory on device execution time : %f ms (%f) \n", elapsed_candidates_memory_dev, elapsed_candidates_memory_dev/elapsed_total);
    printf(" copy candidates to host execution time : %f ms (%f) \n", elapsed_candidates_copy_host, elapsed_candidates_copy_host/elapsed_total);
    printf(" candidates kernel execution time : %f ms (%f) \n", elapsed_candidates_kernel, elapsed_candidates_kernel/elapsed_total);
    printf(" coalesce triggers execution time : %f ms (%f) \n", elapsed_coalesce, elapsed_coalesce/elapsed_total);
    printf(" separate triggers into gates execution time : %f ms (%f) \n", elapsed_gates, elapsed_gates/elapsed_total);
    printf(" deallocate memory execution time : %f ms (%f) \n", elapsed_free, elapsed_free/elapsed_total);
  }
  printf(" total execution time : %f ms \n", elapsed_total);


  return 1;
}




//
// kernel routine
// 

// __global__ identifier says it's a kernel function
__global__ void kernel_correct_times(unsigned int *ct){


  // get unique id for each thread in each block == vertex index
  unsigned int vertex_index = threadIdx.x + blockDim.x*blockIdx.x;

  // skip if thread is assigned to nonexistent vertex
  if( vertex_index >= constant_n_test_vertices ) return;

  //    printf( " vertex_index %d threadidx %d blockdim %d blockid %d \n",
  //	    vertex_index, threadIdx.x, blockDim.x, blockIdx.x);


  unsigned int vertex_block = constant_n_time_bins*vertex_index;

  unsigned int vertex_block2 = constant_n_PMTs*vertex_index;
  for(unsigned int hit_index=0; hit_index<constant_n_hits; hit_index++){
    atomicAdd(&
	      ct[
		 vertex_block 
		 + int(floor(
			     (tex1Dfetch(tex_times,hit_index)
			      - tex1Dfetch(tex_times_of_flight,
					   vertex_block2 
					   + tex1Dfetch(tex_ids,hit_index) - 1
					   )
			      + constant_time_offset)/constant_time_step_size
			     )
		       )
		 ],1);
  }


  return;

}



